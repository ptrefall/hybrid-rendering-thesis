#include "hip/hip_runtime.h"
#include <optix/optix.h>
#include <optix/optix_math.h>

#include "commonStructs.h"

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

struct PerRayData_shadow
{
  float attenuation;
};

rtDeclareVariable(float3,              geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3,              shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(PerRayData_radiance, prd_radiance,     rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,       rtPayload, );
rtDeclareVariable(optix::Ray,          ray,              rtCurrentRay, );
rtDeclareVariable(float,               t_hit,            rtIntersectionDistance, );

rtDeclareVariable(unsigned int,        shadow_ray_type, , );
rtDeclareVariable(float,               scene_epsilon, , );
rtDeclareVariable(rtObject,            top_object, , );
rtDeclareVariable(rtObject,            top_shadower, , );

rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float,  phong_exp, , );
rtDeclareVariable(float3, ambient_light_color, , );
rtBuffer<BasicLight> lights; 

RT_PROGRAM void closest_hit_radiance()
{
	float3 world_geo_normal = normalize(rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal ));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal); // -N or +N
	float3 color = Ka * ambient_light_color;
	float3 hit_point = ray.origin + t_hit * ray.direction;

	for(int i = 0; i < lights.size(); ++i) {
		BasicLight light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = dot( ffnormal, L);

		if( nDl > 0.f ) {
			// cast shadow ray
			PerRayData_shadow prd_shadow;
			prd_shadow.attenuation = 1.0f;
			float Ldist = length(light.pos - hit_point);
			optix::Ray shadow_ray( hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
			rtTrace(top_shadower, shadow_ray, prd_shadow);
			float light_att = prd_shadow.attenuation;

			if ( light_att > 0.0f ) {
				float3 Lc = light.color * light_att;
				color += Kd * nDl * Lc;

				float3 H = normalize( L-ray.direction );
				float nDh = dot( ffnormal, H );
				if ( nDh > 0 ) {
					color += Ks * Lc * pow(nDh, phong_exp);
				}
			}
		}
	}
	prd_radiance.result = color;
}

RT_PROGRAM void any_hit_shadow()
{
	prd_shadow.attenuation = 0.0f; // fully attenuate on opaque...
	rtTerminateRay();
}