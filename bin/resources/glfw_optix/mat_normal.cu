#include "hip/hip_runtime.h"
#include <optix/optix.h>
#include <optix/optix_math.h>
// Used by all the material cuda files
#include "commonStructs.h"

struct PerRayData_radiance
{
  float3 result;
  float  importance;  // This is ignored in this sample.  See phong.h for use.
  int    depth;
};

struct PerRayData_shadow
{
  float3 attenuation;
};

// shading_normal is set by the closest hit intersection program 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

//
// Returns a solid color as the shading result 
// 
RT_PROGRAM void closest_hit_radiance()
{
  prd_radiance.result = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal))*0.5f + 0.5f;
}
