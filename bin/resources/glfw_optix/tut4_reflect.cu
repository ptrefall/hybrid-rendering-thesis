#include "hip/hip_runtime.h"
#include <optix/optix.h>
#include <optix/optix_math.h>

#include "commonStructs.h"

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

struct PerRayData_shadow
{
  float attenuation;
};

rtDeclareVariable(float3,              geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3,              shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(PerRayData_radiance, prd_radiance,     rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,       rtPayload, );
rtDeclareVariable(optix::Ray,          ray,              rtCurrentRay, );
rtDeclareVariable(float,               t_hit,            rtIntersectionDistance, );


rtDeclareVariable(unsigned int,        radiance_ray_type, , );
rtDeclareVariable(unsigned int,        shadow_ray_type, , );
rtDeclareVariable(float,               scene_epsilon, , );
rtDeclareVariable(rtObject,            top_object, , );
rtDeclareVariable(rtObject,            top_shadower, , );

rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float,  phong_exp, , );
rtDeclareVariable(float3, ambient_light_color, , );
rtBuffer<BasicLight> lights; 

// NEW 
rtDeclareVariable(float3, reflectivity, , );
//rtDeclareVariable(float, importance_cutoff, , );
rtDeclareVariable(int, max_depth, , );

RT_PROGRAM void closest_hit_radiance()
{
	float3 world_geo_normal = normalize(rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal ));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal); // -N or +N
	float3 color = Ka * ambient_light_color;
	float3 hit_point = ray.origin + t_hit * ray.direction;

	for(int i = 0; i < lights.size(); ++i) {
		BasicLight light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = dot( ffnormal, L);

		if( nDl > 0.f ) {
			// cast shadow ray
			PerRayData_shadow prd_shadow;
			prd_shadow.attenuation = 1.0f;
			float Ldist = length(light.pos - hit_point);
			optix::Ray shadow_ray( hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
			rtTrace(top_shadower, shadow_ray, prd_shadow);
			float light_att = prd_shadow.attenuation;

			if ( light_att > 0.0f ) {
				float3 Lc = light.color * light_att;
				color += Kd * nDl * Lc;

				float3 H = normalize( L-ray.direction );
				float nDh = dot( ffnormal, H );
				if ( nDh > 0 ) {
					color += Ks * Lc * pow(nDh, phong_exp);
				}
			}
		}
	}

	// Tracking a rays "importance" can improve performance by avoiding
	// creating reflection rays when a color is too dim to have any effect.
	// The luminance function computes a brightness value for the color
	// so we can compute its importance
	float importance = prd_radiance.importance * optix::luminance( reflectivity );

	// reflection ray
	float importance_cutoff = 0.01f;
	if ( importance > importance_cutoff && prd_radiance.depth < max_depth )
	{
		PerRayData_radiance prd_reflect;
		prd_reflect.importance = importance;
		prd_reflect.depth = prd_radiance.depth+1;
		float3 R = reflect( ray.direction, ffnormal );
		optix::Ray refl_ray( hit_point, R, radiance_ray_type, scene_epsilon );
		rtTrace( top_object, refl_ray, prd_reflect );
		color += reflectivity * prd_reflect.result;
	}

	prd_radiance.result = color;
}

RT_PROGRAM void any_hit_shadow()
{
	prd_shadow.attenuation = 0.0f; // fully attenuate on opaque...
	rtTerminateRay();
}