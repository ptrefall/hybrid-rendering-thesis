#include "hip/hip_runtime.h"
#include <optix/optix.h>
#include <optix/optix_math.h>

#include "commonStructs.h"

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

rtDeclareVariable(float3,              geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3,              shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(PerRayData_radiance, prd_radiance,     rtPayload, );
rtDeclareVariable(optix::Ray,          ray,              rtCurrentRay, );
rtDeclareVariable(float,               t_hit,            rtIntersectionDistance, );

rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float,  phong_exp, , );
rtDeclareVariable(float3, ambient_light_color, , );
rtBuffer<BasicLight> lights; 

RT_PROGRAM void closest_hit_radiance()
{
	float3 world_geo_normal = normalize(rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal ));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal); // -N or +N
	float3 color = Ka * ambient_light_color;
	float3 hit_point = ray.origin + t_hit * ray.direction;

	for(int i = 0; i < lights.size(); ++i) {
		BasicLight light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = dot( ffnormal, L);
		if( nDl > 0.f ) {
			float3 Lc = light.color;
			color += Kd * nDl * Lc;
			float3 H = normalize( L-ray.direction );
			float nDh = dot( ffnormal, H );
			if ( nDh > 0.f ) {
				color += Ks * Lc * pow(nDh, phong_exp);
			}
		}
	}
	prd_radiance.result = color;
}