#include "hip/hip_runtime.h"
//#include <optix_world.h>

#include <optix/optix.h>
#include <optix/optix_math.h>

rtDeclareVariable(float3, bg_color, , );

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

RT_PROGRAM void miss()
{
  prd_radiance.result = bg_color;
}
