#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix/optix.h>
#include <optix/optixu/optixu_math_namespace.h>
#include "optix/optixu/optixu_math.h"
#include "helpers.h"

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

struct PerRayData_shadow
{
  float3 attenuation;
};


rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type , , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(rtObject,     top_shadower, , );

//
// Dielectric surface shader from OptiX tutorial 9 (simpler version of what the glass sample uses)
//
rtDeclareVariable(float3,       cutoff_color, , );
rtDeclareVariable(float,        fresnel_exponent, , );
rtDeclareVariable(float,        fresnel_minimum, , );
rtDeclareVariable(float,        fresnel_maximum, , );
rtDeclareVariable(float,        refraction_index, , );
rtDeclareVariable(int,          refraction_maxdepth, , );
rtDeclareVariable(int,          reflection_maxdepth, , );
rtDeclareVariable(float3,       refraction_color, , );
rtDeclareVariable(float3,       reflection_color, , );
rtDeclareVariable(float3,       extinction_constant, , );

rtDeclareVariable(float,    importance_cutoff, , );      
rtDeclareVariable(int,      max_depth, , );

RT_PROGRAM void closest_hit_radiance()
{
  // intersection vectors
  const float3 h = ray.origin + t_hit * ray.direction;            // hitpoint
  const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
  const float3 i = ray.direction;                                            // incident direction

  float reflection = 1.0f;
  float3 result = make_float3(0.0f);

  float3 beer_attenuation;
  if(dot(n, ray.direction) > 0){
    // Beer's law attenuation
    beer_attenuation = optix::expf(extinction_constant * t_hit);
  } else {
    beer_attenuation = make_float3(1);
  }

  // refraction
  if (prd_radiance.depth < min(refraction_maxdepth, max_depth))
  {
    float3 t;                                                            // transmission direction
    if ( refract(t, i, n, refraction_index) )
    {

      // check for external or internal reflection
      float cos_theta = dot(i, n);
      if (cos_theta < 0.0f)
        cos_theta = -cos_theta;
      else
        cos_theta = dot(t, n);

      reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

      float importance = prd_radiance.importance * (1.0f-reflection) * optix::luminance( refraction_color * beer_attenuation );
      if ( importance > importance_cutoff ) {
        optix::Ray ray( h, t, radiance_ray_type, scene_epsilon );
        PerRayData_radiance refr_prd;
        refr_prd.depth = prd_radiance.depth+1;
        refr_prd.importance = importance;

        rtTrace( top_object, ray, refr_prd );
        result += (1.0f - reflection) * refraction_color * refr_prd.result;
      } else {
        result += (1.0f - reflection) * refraction_color * cutoff_color;
      }
    }
    // else TIR
  }

  // reflection
  if (prd_radiance.depth < min(reflection_maxdepth, max_depth))
  {
    float3 r = reflect(i, n);

    float importance = prd_radiance.importance * reflection * optix::luminance( reflection_color * beer_attenuation );
    if ( importance > importance_cutoff ) {
      optix::Ray ray( h, r, radiance_ray_type, scene_epsilon );
      PerRayData_radiance refl_prd;
      refl_prd.depth = prd_radiance.depth+1;
      refl_prd.importance = importance;

      rtTrace( top_object, ray, refl_prd );
      result += reflection * reflection_color * refl_prd.result;
    } else {
      result += reflection * reflection_color * cutoff_color;
    }
  }

  result = result * beer_attenuation;

  prd_radiance.result = result;
}


// -----------------------------------------------------------------------------

//
// Attenuates shadow rays for shadowing transparent objects
//
rtDeclareVariable(float3, shadow_attenuation, , );

RT_PROGRAM void any_hit_shadow()
{
  float3 world_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float nDi = fabs(dot(world_normal, ray.direction));

  prd_shadow.attenuation *= 1-fresnel_schlick(nDi, 5, 1-shadow_attenuation, make_float3(1));
  if(optix::luminance(prd_shadow.attenuation) < importance_cutoff)
    rtTerminateRay();
  else
    rtIgnoreIntersection();
}
