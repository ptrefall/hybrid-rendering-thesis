#include "hip/hip_runtime.h"
#include <Optix/optix.h>
#include <Optix/optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, fTime, , ); 
rtBuffer<float4, 2>   out_buffer;


RT_PROGRAM void sampleTex()
{
	float2 zeroToOne = make_float2(launch_index) / make_float2(launch_dim);
	float2 minusOneToOne = -1.f + 2.f * zeroToOne;
	minusOneToOne *= 5;
	
	out_buffer[launch_index] = make_float4( minusOneToOne.x, minusOneToOne.y, sin( minusOneToOne.x*minusOneToOne.y+3.f*fTime ), 0.f );
}
