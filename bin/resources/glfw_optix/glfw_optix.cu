#include "hip/hip_runtime.h"
#include <Optix/optix.h>
#include <Optix/optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, fTime, , ); 
rtBuffer<uchar4, 2>   out_buffer;

// Convert a float3 in [0,1)^3 to a uchar4 in [0,255]^4 -- 4th channel is set to 255
#ifdef __HIPCC__
__device__ __inline__ optix::uchar4 make_color(const optix::float3& c)
{
    return optix::make_uchar4( static_cast<unsigned char>(__saturatef(c.z)*255.99f),  /* B */
                               static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
                               static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* R */
                               255u);                                                 /* A */
}
#endif

RT_PROGRAM void sampleTex()
{
	float2 zeroToOne = make_float2(launch_index) / make_float2(launch_dim);
	float2 minusOneToOne = -1.f + 2.f * zeroToOne;
	minusOneToOne *= 5;
	
	out_buffer[launch_index] = make_color( make_float3( minusOneToOne.x, minusOneToOne.y, sin( minusOneToOne.x*minusOneToOne.y+3.f*fTime ) ) );
}
