#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <Optix/optix.h>
#include <Optix/optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, fTime, , ); 
rtBuffer<float4, 2>   out_buffer;
rtTextureSampler<float4, 2> tex;

RT_PROGRAM void sampleTex()
{
	float2 zeroToOne = make_float2(launch_index) / make_float2(launch_dim);
	//float2 minusOneToOne = make_float2( -1.f + 2.f * zeroToOne.x, -1.f + 2.f * zeroToOne.y );
	float2 minusOneToOne = -1.f + 2.f * zeroToOne;
	minusOneToOne *= 5;
	//out_buffer[launch_index] = tex2D( tex, zeroToOne.x, zeroToOne.y );
	
	//out_buffer[launch_index] = tex2D( tex, zeroToOne.x, zeroToOne.y );

	out_buffer[launch_index] = make_float4( minusOneToOne.x, minusOneToOne.y, sin( minusOneToOne.x*minusOneToOne.y+3.f*fTime ), 0.f );

	out_buffer[launch_index].x = 255 - out_buffer[launch_index].x;
	out_buffer[launch_index].y = 255 - out_buffer[launch_index].y;
	out_buffer[launch_index].z = 255 - out_buffer[launch_index].z;
	out_buffer[launch_index].w = 255;
}
