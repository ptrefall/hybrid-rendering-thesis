#include "hip/hip_runtime.h"
#include <optix/optix.h>
#include <optix/optix_math.h>
rtDeclareVariable(float3,          background_light, , ); // horizon color
rtDeclareVariable(float3,          background_dark, , );  // zenith color
rtDeclareVariable(float3,          up, , );               // global up vector
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

RT_PROGRAM void miss()
{
	const float t = 0.5f + 0.5f*dot(ray.direction,up); // +0.5 * 0.5 Gives a longer range... might be interesting to mix 3 colors, sky/north-pole, horizon, void/south-pole
	//const float t = max(dot(ray.direction,up), 0.0f);
	prd_radiance.result = lerp(background_light, background_dark,t);
}
