#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include "optix.h"
#include "helpers.h"
#include "optixu/optixu_math_namespace.h"

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtBuffer<uchar4, 2>   g_buffer; // Map this to a PBO

rtTextureSampler<float4, 2> raster_diffuse_tex;
rtTextureSampler<float4, 2> raster_position_tex;
rtTextureSampler<float4, 2> raster_normal_tex;


RT_PROGRAM void gbuffer_compose()
{ 
	//g_buffer[launch_index].x = 255 - g_buffer[launch_index].x;
	//g_buffer[launch_index].y = 255 - g_buffer[launch_index].y;
	//g_buffer[launch_index].z = 255 - g_buffer[launch_index].z;
	//g_buffer[launch_index].w = 255;
	//float2 zeroToOne = make_float2(launch_index) / make_float2(launch_dim);
	//g_buffer[launch_index] = make_color( make_float3( zeroToOne.x, zeroToOne.y, 0.f ) );
	g_buffer[launch_index] = make_color( make_float3( 255.0f, 255.0f, 255.0f ) );
	//result_buffer[launch_index] = tex2D( tex, zeroToOne.x, zeroToOne.y );


	/*
	float4 diffuse = ( tex2D( raster_diffuse_tex, uv.x, uv.y ) );
	float4 position = ( tex2D( raster_position_tex, uv.x, uv.y ) );
	float4 normal_matid = ( tex2D( raster_normal_tex, uv.x, uv.y ) );

	result_buffer[launch_index] = shade...
	*/
}

