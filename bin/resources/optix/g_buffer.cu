#include "hip/hip_runtime.h"


#include "optix.h"
#include "helpers.h"
#include "optixu/optixu_math_namespace.h"

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtBuffer<uchar4, 2>   g_buffer; // Map this to a PBO

//rtTextureSampler<float4, 2> raster_diffuse_tex;
//rtTextureSampler<float4, 2> raster_position_tex;
//rtTextureSampler<float4, 2> raster_normal_tex;

// Convert a float3 in [0,1)^3 to a uchar4 in [0,255]^4 -- 4th channel is set to 255
#ifdef __HIPCC__
__device__ __inline__ optix::uchar4 make_color(const optix::float3& c)
{
    return optix::make_uchar4( static_cast<unsigned char>(__saturatef(c.z)*255.99f),  /* B */
                               static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
                               static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* R */
                               255u);                                                 /* A */
}
#endif

RT_PROGRAM void gbuffer_compose()
{ 
	//g_buffer[launch_index].x = 255 - g_buffer[launch_index].x;
	//g_buffer[launch_index].y = 255 - g_buffer[launch_index].y;
	//g_buffer[launch_index].z = 255 - g_buffer[launch_index].z;
	//g_buffer[launch_index].w = 255;
	//float2 zeroToOne = make_float2(launch_index) / make_float2(launch_dim);
	//g_buffer[launch_index] = make_color( make_float3( zeroToOne.x, zeroToOne.y, 0.f ) );
	g_buffer[launch_index] = make_color( make_float3( 255.0f, 0.0f, 0.0f ) );

	/*
	float4 diffuse = ( tex2D( raster_diffuse_tex, uv.x, uv.y ) );
	float4 position = ( tex2D( raster_position_tex, uv.x, uv.y ) );
	float4 normal_matid = ( tex2D( raster_normal_tex, uv.x, uv.y ) );

	result_buffer[launch_index] = shade...
	*/
}

