#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include "optix.h"
#include "optixu/optixu_math_namespace.h"

using namespace optix;

struct PerRayData_tex
{
  float3 diffuse;
  float3 position;
  float3 normal_matid;
};

rtTextureSampler<float4, 2> raster_diffuse_tex;
rtTextureSampler<float4, 2> raster_position_tex;
rtTextureSampler<float4, 2> raster_normal_tex;
rtDeclareVariable(PerRayData_tex, prd, rtPayload, );
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 

RT_PROGRAM void closest_hit_radiance()
{
  const float3 uv = texcoord;

  prd.diffuse = make_float3( tex2D( raster_diffuse_tex, uv.x, uv.y ) );
  prd.position = make_float3( tex2D( raster_position_tex, uv.x, uv.y ) );
  prd.normal_matid = make_float3( tex2D( raster_normal_tex, uv.x, uv.y ) );
}
