#include "hip/hip_runtime.h"


#include "optix.h"
#include "helpers.h"
#include "optixu/optixu_math_namespace.h"

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtBuffer<uchar4, 2>   g_buffer_diffuse_read;
rtBuffer<float4, 2>   g_buffer_position_read;
rtBuffer<float4, 2>   g_buffer_normal_read;

rtBuffer<uchar4, 2>   g_buffer_diffuse_write;
rtBuffer<float4, 2>   g_buffer_position_write;
rtBuffer<float4, 2>   g_buffer_normal_write;

//rtTextureSampler<float4, 2> raster_diffuse_tex;
//rtTextureSampler<float4, 2> raster_position_tex;
//rtTextureSampler<float4, 2> raster_normal_tex;

RT_PROGRAM void gbuffer_compose()
{ 
	//g_buffer_diffuse[launch_index].x = g_buffer_diffuse[launch_index].x;
	//g_buffer_diffuse[launch_index].y = g_buffer_diffuse[launch_index].y;
	//g_buffer_diffuse[launch_index].z = g_buffer_diffuse[launch_index].z;
	//g_buffer_diffuse[launch_index].w = g_buffer_diffuse[launch_index].w;

	//g_buffer_position[launch_index].x = g_buffer_position[launch_index].x;
	//g_buffer_position[launch_index].y = g_buffer_position[launch_index].y;
	//g_buffer_position[launch_index].z = g_buffer_position[launch_index].z;
	//g_buffer_position[launch_index].w = g_buffer_position[launch_index].w;

	//g_buffer_normal[launch_index].x = g_buffer_normal[launch_index].x;
	//g_buffer_normal[launch_index].y = g_buffer_normal[launch_index].y;
	//g_buffer_normal[launch_index].z = g_buffer_normal[launch_index].z;
	//g_buffer_normal[launch_index].w = g_buffer_normal[launch_index].w;
	
	//float2 zeroToOne = make_float2(launch_index) / make_float2(launch_dim);
	//g_buffer[launch_index] = make_color( make_float3( zeroToOne.x, zeroToOne.y, 0.f ) );
	//g_buffer[launch_index] = make_color( make_float3( 255.0f, 0.0f, 0.0f ) );

	/*
	float4 diffuse = ( tex2D( raster_diffuse_tex, uv.x, uv.y ) );
	float4 position = ( tex2D( raster_position_tex, uv.x, uv.y ) );
	float4 normal_matid = ( tex2D( raster_normal_tex, uv.x, uv.y ) );

	result_buffer[launch_index] = shade...
	*/
}

