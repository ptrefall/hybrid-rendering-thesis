#include "hip/hip_runtime.h"
#include "optix_world.h"

rtDeclareVariable(float3, bg_color, , );

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

RT_PROGRAM void miss()
{
  prd_radiance.result = make_float3(0.f,0.f,1.f);
  prd_radiance.result = bg_color;
}
