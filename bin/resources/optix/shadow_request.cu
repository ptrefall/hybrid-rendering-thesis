#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

//rtBuffer<uchar4, 2>   g_buffer_diffuse_read;
rtBuffer<float4, 2>   g_buffer_position_read;
//rtBuffer<float4, 2>   g_buffer_normal_read;

rtBuffer<uchar4, 2>   g_buffer_diffuse_write;
//rtBuffer<float4, 2>   g_buffer_position_write;
//rtBuffer<float4, 2>   g_buffer_normal_write;

rtDeclareVariable(uint, shadow_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

//rtDeclareVariable(float3, light_pos, , );
rtDeclareVariable(rtObject, top_object, , );

struct PerRayData_shadow
{
  float attenuation;
};

RT_PROGRAM void shadow_request()
{
  //float3 ray_origin = make_float3(tex2D(g_buffer_position, launch_index.x, launch_index.y));
  float3 ray_origin = make_float3( g_buffer_position_read[launch_index] ) ; // xyz
  

  PerRayData_shadow prd_shadow;
  prd_shadow.attenuation = 1.0f;

  if( !isnan(ray_origin.x) ) {
	float3 light_pos = make_float3( 2.35f,2.00f,1.10f );
    float3 L = light_pos-ray_origin;
    float dist = sqrtf(dot(L,L));
    float3 ray_direction = L/dist;
    optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, shadow_ray_type, scene_epsilon, dist);
    rtTrace(top_object, ray, prd_shadow);
  }

 //g_buffer_diffuse_write[launch_index].x = static_cast<unsigned char>(prd_shadow.attenuation*255.99f);

  // G-buffer Position debugging with limited range...
  //g_buffer_diffuse_write[launch_index].x = static_cast<unsigned char>(ray_origin.x*255.99f);
  //g_buffer_diffuse_write[launch_index].y = static_cast<unsigned char>(ray_origin.y*255.99f);
  //g_buffer_diffuse_write[launch_index].z = static_cast<unsigned char>(-ray_origin.z*255.99f);
}