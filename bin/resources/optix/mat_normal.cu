#include "hip/hip_runtime.h"
#include <optix/optix.h>
#include <optix/optix_math.h>

struct PerRayData_radiance
{
  float3 result;
  float  importance;  // This is ignored in this sample.  See phong.h for use.
  int    depth;
};

struct PerRayData_shadow
{
  float attenuation;
};

// shading_normal is set by the closest hit intersection program 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

RT_PROGRAM void closest_hit_radiance()
{
	prd_radiance.result = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal))*0.5f + 0.5f;
}

RT_PROGRAM void any_hit_radiance()
{
	prd_shadow.attenuation = 0.f;
}