#include "hip/hip_runtime.h"


#include <Optix/optix_world.h>
#include "helpers.h"

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float  importance;
  int    depth;
};

struct PerRayData_shadow
{
  float attenuation;
};

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );

//rtBuffer<uchar4, 2>   g_buffer_diffuse_read;
rtBuffer<float4, 2>   g_buffer_position_read;
//rtBuffer<float4, 2>   g_buffer_normal_read;

rtBuffer<uchar4, 2>   g_buffer_diffuse_write;
//rtBuffer<float4, 2>   g_buffer_position_write;
//rtBuffer<float4, 2>   g_buffer_normal_write;

rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

RT_PROGRAM void pinhole_camera()
{
  float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);
  
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  g_buffer_diffuse_write[launch_index] = make_color( prd.result );
  
  // Copy
  //g_buffer_position_write[launch_index] = g_buffer_position_read[launch_index];
  //g_buffer_normal_write[launch_index] = g_buffer_normal_read[launch_index];
}
