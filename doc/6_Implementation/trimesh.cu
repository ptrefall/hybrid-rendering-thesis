#include "hip/hip_runtime.h"

rtBuffer<float3>              vertex_buffer;     
rtBuffer<int3>                index_buffer;    // position indices

rtDeclareVariable(float3,     texcoord,         attribute texcoord, ); 
rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(int,        normal_offset,    , ); 
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );

RT_PROGRAM void mesh_intersect( int primIdx )
{
	int3 v_idx = index_buffer[primIdx];

	float3 p0 = vertex_buffer[ v_idx.x ];
	float3 p1 = vertex_buffer[ v_idx.y ];
	float3 p2 = vertex_buffer[ v_idx.z ];

	// Intersect ray with triangle
	float3 n;
	float  t, beta, gamma;

	if( intersect_triangle( ray, p0, p1, p2, n, t, beta, gamma ) ) {
	if ( dot(ray.direction, n) < 0 ) // Don't render backfaces
	if(  rtPotentialIntersection( t ) ) {

	geometric_normal = normalize( n );

	if ( normal_offset <= 0 ) {
			shading_normal   = geometric_normal;
		} else {
			float3 n0 = vertex_buffer[ v_idx.x+normal_offset ];
			float3 n1 = vertex_buffer[ v_idx.y+normal_offset ];
			float3 n2 = vertex_buffer[ v_idx.z+normal_offset ];
			shading_normal = 
			normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
		}

		texcoord = make_float3( 0.0f, 0.0f, 0.0f );

		rtReportIntersection( 0 );
	}
	}
}
